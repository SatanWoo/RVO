#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "AddVector.cuh"

__global__ void addKernelI(float32 *c, float32 *a, float32 *b, int size)
{
	int ID = threadIdx.x;
	if (ID < size) {
		c[ID] = a[ID] + b[ID];
	}
}

__global__ void addKernelC(float32 *c, float32 k, int size)
{
	int ID = threadIdx.x;
	if (ID < size) {
		c[ID] = k;
	}
}

__global__ void addKernelV(b2Vec2 *c, b2Vec2 vec, int size) 
{
	int ID = threadIdx.x;
	if (ID < size) {
		c[ID] = vec;
	}
}

void addVectorI(float32 *re, float32 *a, float32 *b, int size)
{
	float32 *dev_a = 0;
	float32 *dev_b = 0;
	float32 *dev_c = 0;

	hipSetDevice(0);

	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));

	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	addKernelI<<<1, size>>>(dev_c, dev_a, dev_b, size);

	hipDeviceSynchronize();
	hipMemcpy(re, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}

void addVectorC(float32 *c, float32 k, int size)
{
	float32 *dev_c = 0;

	hipSetDevice(0);
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);

	addKernelC<<<1, size>>>(dev_c, k, size);

	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);
}

void addVectorV(b2Vec2 *c, b2Vec2 vec, int size)
{
	b2Vec2 *dev_c = 0;

	hipSetDevice(0);
	hipMalloc((void**)&dev_c, size * sizeof(b2Vec2));
	hipMemcpy(dev_c, c, size * sizeof(b2Vec2), hipMemcpyHostToDevice);

	addKernelV<<<1, size>>>(dev_c, vec, size);

	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, size * sizeof(b2Vec2), hipMemcpyDeviceToHost);

	hipFree(dev_c);
}